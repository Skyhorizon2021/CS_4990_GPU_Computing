#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16

#define CHECK(call)                                           \
{                                                             \
    const hipError_t error = call;                           \
    if (error != hipSuccess)                                 \
    {                                                         \
        printf("Error: %s:%d, ", __FILE__, __LINE__);         \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(1);                                              \
    }                                                         \
}

__global__ void gpu_matrix_mult(int *a, int *b, int *c, int m, int n, int k)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if (col < k && row < m) 
    {
        for (int i = 0; i < n; i++) 
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
} 

__global__ void gpu_square_matrix_mult(int *d_a, int *d_b, int *d_result, int n) 
{
    __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int tmp = 0;
    int idx;

    for (int sub = 0; sub < gridDim.x; ++sub) 
    {
        idx = row * n + sub * BLOCK_SIZE + threadIdx.x;
        if (idx >= n * n)
        {
            tile_a[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            tile_a[threadIdx.y][threadIdx.x] = d_a[idx];
        }

        idx = (sub * BLOCK_SIZE + threadIdx.y) * n + col;
        if (idx >= n * n)
        {
            tile_b[threadIdx.y][threadIdx.x] = 0;
        }  
        else
        {
            tile_b[threadIdx.y][threadIdx.x] = d_b[idx];
        }
        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k) 
        {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }
        __syncthreads();
    }
    if (row < n && col < n)
    {
        d_result[row * n + col] = tmp;
    }
}

void cpu_matrix_mult(int *h_a, int *h_b, int *h_result, int m, int n, int k) {
    for (int i = 0; i < m; ++i) 
    {
        for (int j = 0; j < k; ++j) 
        {
            int tmp = 0; // Use int type for tmp
            for (int h = 0; h < n; ++h) 
            {
                tmp += h_a[i * n + h] * h_b[h * k + j];
            }
            h_result[i * k + j] = tmp;
        }
    }
}

int main(int argc, char const *argv[])
{
    int m, n, k;
    srand(3333);
    printf("please type in m n and k\n");
    scanf("%d %d %d", &m, &n, &k);

    // Allocate memory in host RAM, h_cc is used to store CPU result
    int *h_a, *h_b, *h_c, *h_cc;
    CHECK(hipHostMalloc((void **) &h_a, sizeof(int) * m * n, hipHostMallocDefault));
    CHECK(hipHostMalloc((void **) &h_b, sizeof(int) * n * k, hipHostMallocDefault));
    CHECK(hipHostMalloc((void **) &h_c, sizeof(int) * m * k, hipHostMallocDefault));
    CHECK(hipHostMalloc((void **) &h_cc, sizeof(int) * m * k, hipHostMallocDefault));

    // Random initialize matrix A
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            h_a[i * n + j] = rand() % 100;
        }
    }

    // Random initialize matrix B
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < k; ++j) {
            h_b[i * k + j] = rand() % 100;
        }
    }

    float gpu_elapsed_time_ms, cpu_elapsed_time_ms;

    // Events to count the execution time
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    // Start to count execution time of GPU version
    CHECK(hipEventRecord(start, 0));
    
    // Allocate memory space on the device 
    int *d_a, *d_b, *d_c;
    CHECK(hipMalloc((void **) &d_a, sizeof(int) * m * n));
    CHECK(hipMalloc((void **) &d_b, sizeof(int) * n * k));
    CHECK(hipMalloc((void **) &d_c, sizeof(int) * m * k));

    // Copy matrix A and B from host to device memory
    CHECK(hipMemcpy(d_a, h_a, sizeof(int) * m * n, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, h_b, sizeof(int) * n * k, hipMemcpyHostToDevice));

    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
   
    // Launch kernel 
    if (m == n && n == k)
    {
        gpu_square_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, n);    
    }
    else
    {
        gpu_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m, n, k);    
    }
    
    CHECK(hipDeviceSynchronize());

    // Transfer results from device to host 
    CHECK(hipMemcpy(h_c, d_c, sizeof(int) * m * k, hipMemcpyDeviceToHost));

    // Time counting terminate
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));

    // Compute time elapsed on GPU computing
    CHECK(hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop));
    printf("%dx%d . %dx%d on GPU: %f ms.\n\n", m, n, n, k, gpu_elapsed_time_ms);

    // Start the CPU version
    CHECK(hipEventRecord(start, 0));

    cpu_matrix_mult(h_a, h_b, h_cc, m, n, k);

    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&cpu_elapsed_time_ms, start, stop));
    printf("%dx%d . %dx%d on CPU: %f ms.\n\n", m, n, n, k, cpu_elapsed_time_ms);

    // Validate results computed by GPU
    int all_ok = 1;
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            if (h_cc[i * k + j] != h_c[i * k + j])
            {
                all_ok = 0;
                break;
            }
        }
        if (!all_ok) break;
    }

    // Compute speedup
    if (all_ok)
    {
        printf("Speedup = %f\n", cpu_elapsed_time_ms / gpu_elapsed_time_ms);
    }
    else
    {
        printf("Incorrect results\n");
    }

    // Free memory
    CHECK(hipFree(d_a));
    CHECK(hipFree(d_b));
    CHECK(hipFree(d_c));
    CHECK(hipHostFree(h_a));
    CHECK(hipHostFree(h_b));
    CHECK(hipHostFree(h_c));
    CHECK(hipHostFree(h_cc));

    return 0;
}
